#include "hip/hip_runtime.h"
// Alex O., May 1st, 2011
// main function for matmul program

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matmul.h"

int main()
{
	int i,j,k;
	Matrix A, B, C, Cref;
	int correct = 1;

	// Allocate and initialize A, B	and C
	A.width = W;
	A.height = H;
	A.elements = (float*)malloc(A.width*A.height*sizeof(float));
	for (i=0;i<A.height;i++)
		for (j=0;j<A.width;j++)
			A.elements[i*A.width+j] = i*3+j;
	B.width = W;
	B.height = H;
	B.elements = (float*)malloc(B.width*B.height*sizeof(float));
	for (i=0;i<B.height;i++)
		for (j=0;j<B.width;j++)
			B.elements[i*B.width+j] = i*2+j;
	C.width = W;
	C.height = H;
	C.elements = (float*)malloc(C.width*C.height*sizeof(float));
	for (i=0;i<A.height;i++)
		for (j=0;j<A.width;j++)
			C.elements[i*C.width+j] = 0;

	printf("Computing multiplication on CPU\n");
	Cref.width = W;
	Cref.height = H;
	Cref.elements = (float*)malloc(Cref.width*Cref.height*sizeof(float));
	for (i=0;i<A.height;i++)
		for (j=0;j<B.width;j++) {
			Cref.elements[i*Cref.width+j]=0;
			for (k=0;k<A.height;k++)
				Cref.elements[i*Cref.width+j] += A.elements[i*A.width+k]*B.elements[k*B.width+j];
		}
	
	printf("Computing multiplication on GPU\n");
	MatMul(A,B,C);			
	printf("Checking result\n");
	for (i=0;i<C.height;i++)
		for(j=0;j<C.width;j++)
		{
#if DEBUG
			printf("%f %f\n", C.elements[i*C.width+j]+0, Cref.elements[i*Cref.width+j]);
#endif
			if (isnan(C.elements[i*C.width+j]) || abs(C.elements[i*C.width+j]-Cref.elements[i*Cref.width+j])>0.01)
				correct = 0;
		}

	if (correct)
		printf("PASSED\n");
	else
		printf("FAILED\n");

	free(A.elements);
	free(B.elements);
	free(C.elements);
	return 0;

}
