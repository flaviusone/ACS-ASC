#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector dot product: C = A + B.
 *
 * This sample is a very basic sample that implements
 * vector dot product. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
/**
 * Host main routine
 */


float make_rand()
{
	return rand()/(float)RAND_MAX;
}

int main(int argc, char **argv)
{

    int numElements = 1024;
    printf("[Vector dot product of %d elements]\n", numElements);

    thrust::host_vector<float> h_A(numElements);
    thrust::host_vector<float> h_B(numElements);

    srand(time(NULL));
    thrust::generate(h_A.begin(), h_A.end(), make_rand);
    thrust::generate(h_B.begin(), h_B.end(), make_rand);

    //TODO: delarați și inițializați vectorii de pe device d_A, d_B
    thrust::device_vector<float> d_A = h_A;
    thrust::device_vector<float> d_B = h_B;
    //TODO: delarați vectoruld e pe device d_C
    thrust::device_vector<float> d_C(numElements);
    //TODO: calculați produsul element cu element C[i] = A[i] * B[i] folosind transformări
    thrust::transform(d_A.begin(), d_A.end(), d_B.begin(), d_C.begin(), thrust::multiplies<float>() );
    //TODO: calculați suma elementelor lui C în dotProd_d folosind reduceri
    float dotProd_d = thrust::reduce(d_C.begin(), d_C.end(), (float) 0, thrust::plus<float>());

    // Verify that the result vector is correct
    float result = 0;
    for (int i = 0; i < numElements; ++i)
    {
	result += (h_A[i]*h_B[i]);
    }

    printf("CPU = %f\n", result);
    printf("GPU = %f\n", dotProd_d);

    if (fabs(result - dotProd_d) > 1e-2)
    {
	fprintf(stderr, "Result verification failed!\n");
    }

    printf("Done\n");
    return 0;
}

