#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector dot product: C = A + B.
 *
 * This sample is a very basic sample that implements
 * vector dot product. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

template <typename T>
struct square
{
	//TODO: definiți opearatorul care calculează pătratul unei valori
    __host__ __device__ T operator() (const T &a) const {
        return a * a;
    }
};

/**
 * Host main routine
 */


float make_rand()
{
	return rand()/(float)RAND_MAX;
}

int main(int argc, char **argv)
{

    // Print the vector length to be used, and compute its size
    int numElements = 1024;
    printf("[Reduce vector of %d elements]\n", numElements);

    thrust::host_vector<float> h_A(numElements);

    square<float> unary_op;
    thrust::plus<float> plus;
    float init = 0.0;

    float res_d = 0.0, res_h = 0.0;

    srand(time(NULL));
    thrust::generate(h_A.begin(), h_A.end(), make_rand);

    //TODO: declarați și inițializați vectorul d_A de pe device
    thrust::device_vector<float> d_A = h_A;
    //TODO: declarați vectorul C de lungime numElements pe device
    thrust::device_vector<float> d_C(numElements);
    //TODO: calculați folosind transform_reduce suma pătratelor elementelor lui d_A în res_d
    res_d = thrust::transform_reduce(d_A.begin(), d_A.end(), square<float>(), init, plus);

    //TODO: calculați folosind transform_reduce suma pătratelor elementelor lui h_A în res_h
    res_h = thrust::transform_reduce(h_A.begin(), h_A.end(), square<float>(), init, plus);


    printf("CPU = %f\n", res_h);
    printf("GPU = %f\n", res_d);

    if (fabs(res_h - res_d) > 1e-2)
    {
	fprintf(stderr, "Result verification failed!\n");
    }

    printf("Done\n");
    return 0;
}

