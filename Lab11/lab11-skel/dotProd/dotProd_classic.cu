#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector dot product: C = A + B.
 *
 * This sample is a very basic sample that implements
 * vector dot product. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector dot product of A and B. The vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorDot(const float *A, const float *B, float *C, float *dotProd, int numElements)
{
  int j;
  //TODO: calculați indexul threadului
  int thread_no = threadIdx.x + blockDim.x * blockIdx.x;
  //TODO: fiecare thread face calculul
  if (thread_no < numElements) {
        C[thread_no] = A[thread_no] * B[thread_no];
  }
   __syncthreads();
  //TODO: unul din threaduri calculează suma finală și o pune în dotProduct
  if (thread_no == 0) {
      float suma = 0;
      for ( j = 0; j < numElements; j++)
          suma += C[j];
      dotProd[0] = suma;
  }
}

/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

    // Print the vector length to be used, and compute its size
    int numElements = 1024;
    size_t size = numElements * sizeof(float);

    printf("[Vector dot product of %d elements]\n", numElements);

    float *h_A, *h_B, *h_C, *dotProd_h;
    //TODO: alocați cei 3 vectori h_A, h_B, h_C pe host și dotProd_h care va reține suma finală
    h_A = (float *) malloc(size);
    h_B = (float *) malloc(size);
    h_C = (float *) malloc(size);

    //TODO: inițializați random cei doi vectori h_A si h_B
    srand(time(NULL));
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / 10;
        h_B[i] = rand() / 10;
    }
    //TODO: Alocați vectorii d_A, d_B și d_C pe device și dotProd_d car va reține rezultatul sumei pe device
    float *d_A, *d_B, *d_C, *dotProd_d;

    hipMalloc( (void**) &d_A, size);
    hipMalloc( (void**) &d_B, size);
    hipMalloc( (void**) &d_C, size);
    hipMalloc( (void**) &dotProd_d, sizeof(float));
    dotProd_h = (float *) malloc (sizeof (float));
   //TODO: Copiați datele în d_A și d_B pe device din h_A și h_B
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);
    int threadsPerBlock = 32;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, dotProd_d, numElements);

    //TODO: copiați datele de pe device pe host
    hipMemcpy(dotProd_h, dotProd_d, sizeof(float), hipMemcpyDeviceToHost);


    // Verify that the result vector is correct
    float result = 0;
    for (int i = 0; i < numElements; ++i)
    {
	result += (h_A[i]*h_B[i]);
    }

    printf("CPU = %f\n", result);
    printf("GPU = %f\n", dotProd_h[0]);

    if (fabs(result - dotProd_h[0]) > 1e-5)
    {
	fprintf(stderr, "Result verification failed!\n");
    }


    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(dotProd_d);


    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(dotProd_h);

    printf("Done\n");
    return 0;
}

