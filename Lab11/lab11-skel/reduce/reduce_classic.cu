#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector dot product: C = A + B.
 *
 * This sample is a very basic sample that implements
 * vector dot product. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector dot product of A and B. The vectors have the same
 * number of elements numElements.
 */


__global__ void
reduce(const float *A, float *C, float *res, int numElements)
{
    int j;
   //TODO: calculați indexul threadului
   int no_thread = threadIdx.x + blockDim.x * blockIdx.x;
   //TODO: fiecare thread calculează în C pătratul elementului corespunzător din A
   if (no_thread < numElements) {
        C[no_thread] = A[no_thread] * A[no_thread];
   }
   __syncthreads();
   //TODO: unul dintre threaduri face suma și o returnează în res
    if (no_thread == 0) {
        float sum = 0;
        for (j = 0; j < numElements; ++j)
            sum += C[j];
        res[0] = sum;
    }
}

/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

    // Print the vector length to be used, and compute its size
    int numElements = 1024;
    size_t size = numElements * sizeof(float);
    printf("[Reduce vector of %d elements]\n", numElements);

    // Allocați vectorii h_A, h_B, h_C de pe host si pointerul ce reține rezultatul h_res
    float *h_A, *h_C, *h_res;
    h_A = (float *) malloc(size);
    h_C = (float *) malloc(size);
    h_res = (float *) malloc (sizeof(float));
    //TODO: inițialiați vectorul h_A de pe host
    srand(time(NULL));
    for (int i = 0; i < numElements; i++) {
        h_A[i] = rand() / 100;
    }
    float *d_A, *d_C, *d_res;
    //TODO: alocați spațiu pe device pt cei 2 vectori A și C și pentru pointerul d_res ce reține rezultatul
    hipMalloc( (void**) &d_A, size);
    hipMalloc( (void**) &d_C, size);
    hipMalloc( (void**) &d_res, sizeof(float));
    //TODO: copiați din h_A în d_A
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Launch the Vector Dot Product CUDA Kernel
    int threadsPerBlock = 32;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    reduce<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, d_res, numElements);

    //TODO: copiați rezultatul de pe device (d_res) pe host (h_res)
    hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    // Verify that the result vector is correct
    float result = 0;
    for (int i = 0; i < numElements; ++i)
    {
	result += (h_A[i]*h_A[i]);
    }

    printf("CPU = %f\n", result);
    printf("GPU = %f\n", h_res[0]);

    if (fabs(result - h_res[0]) > 1e-5)
    {
	fprintf(stderr, "Result verification failed!\n");
    }


    // TODO: eliberați memoria ocupată de d_A, d_C și d_res pe device
    hipFree(d_A);
    hipFree(d_C);
    hipFree(d_res);

    free(h_A);
    free(h_C);
    free(h_res);

    printf("Done\n");
    return 0;
}

