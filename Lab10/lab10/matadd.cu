#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define MAX_ERR 0.0001
#define N 10000
#define BS 2

void matAddOnHost(float *a, float *b, float *c){
    int i, j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            c[i * N + j] = a[i * N + j] + b[i * N + j];
        }
    }
}

__global__ void MatAdd(float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < N && j < N) {
        C[i*N+j] = A[i*N+j] + B[i*N+j];
    }
}

int main() {
    int i = 0;
    float a[N*N], b[N*N], c[N*N];
    float *A, *B, *C;
    float *pgpu = (float*) malloc(N*N*sizeof(float));

    /* Umplem matricea cu elemente random*/
    srand((unsigned int) time((time_t *)NULL));
    for(i = 0; i < N * N; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }
    /* Adunam pe host */
    matAddOnHost(a, b, c);


    /* Alocam spatiu pt matrice*/
    hipMalloc((void**) &A, N * N * sizeof(float));
    hipMalloc((void**) &B, N * N * sizeof(float));
    hipMalloc((void**) &C, N * N * sizeof(float));

    /* Le umplem */
    hipMemcpy(A, a, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, b, N*N*sizeof(float), hipMemcpyHostToDevice);


    /* Magic happens*/
    dim3 dimBlock(BS, BS);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    MatAdd<<<dimGrid, dimBlock>>>(A, B, C);

    /* Copy result to host*/
    hipMemcpy(pgpu, C, N*N*sizeof(float), hipMemcpyDeviceToHost);

    /* Free space */
    hipFree(A);
    hipFree(B);
    hipFree(C);

    /* Check if passed*/
    int correct = 1;
    for(i = 0; i < N*N; i++) {
        if (abs(c[i] - pgpu[i])>MAX_ERR) correct = 0;
    }
    if(correct) {
        printf("PASSED\n");
    } else {
        printf("FAILED\n");
    }
    free(pgpu);
}
