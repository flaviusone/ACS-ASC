#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define MAX_ERR 0.0001
#define N 10000
#define BS 2

void doubleArrayOnHost (float* s, float* d, int size)
{
	int i;
	for ( i=0; i<size; i++) d[i] = 2*s[i];
}

__global__ void doubleArrayOnDevice(float* temp, int size)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index<N) temp[index] = 2*temp[index];
}

int main()
{
	int i=0;
	int correct = 1;
	float* p1 = (float*) malloc( N *sizeof(float) );	// original data source
	float* p2 = (float*) malloc( N *sizeof(float) ); 	// holds result computed by CPU
	float* pgpu = (float*) malloc( N *sizeof(float) ); 	// holds result obtained from GPU
	float* pd = 0; 						// pointer to device memory

	// initialize array values
	srand((unsigned int)time((time_t *)NULL));
	for ( i=0; i<N; i++) p1[i] = rand()/10;
	// compute the result on the host
	doubleArrayOnHost(p1, p2, N);

	// compute the result on device
	hipMalloc( (void**) &pd, N *sizeof(float)); 	// the second parameter is the addres of the pointer, not the pointer itself
	hipMemcpy( pd, p1, N*sizeof(float), hipMemcpyHostToDevice);
	int bs = BS; // runtime chosen value for blocksize! 

	// if N is not a multiple of bs, 
	// an extra block is needed
	int nb = N / bs + (N%bs == 0 ? 0:1); 
	doubleArrayOnDevice <<< nb, bs >>> (pd, N);

	hipMemcpy( pgpu, pd, N*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(pd);

	// check consistency
	for ( i=0; i<N; i++)
	{
		if (abs(p2[i] - pgpu[i])>MAX_ERR) correct = 0;
	}
	if (correct) {
		printf("PASSED\n");
	} else {
		printf("FAILED\n");
	}
	free(p1); free(p2); free(pgpu);
}
