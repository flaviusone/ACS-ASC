#include "hip/hip_runtime.h"
#include "matmul.h"
 
// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width; 
  d_A.height = A.height;
  
  size_t size = A.width * A.height * sizeof(float);
  // TODO: alocati matricea d_A pe device si puneti in ea date din matricea A de pe host
  hipMalloc((void **) &d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  Matrix d_B;

  d_B.width = B.width; 
  d_B.height = B.height;
  
  size = B.width * B.height * sizeof(float);
  // TODO: alocati matricea d_B pe device si puneti in ea date din matricea B de pe host
  hipMalloc((void **) &d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);


  // Allocate C in device memory
  Matrix d_C;

  d_C.width = C.width; 
  d_C.height = C.height;
  
  size = C.width * C.height * sizeof(float);
  hipMalloc((void**)&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  // TODO: copiati in matricea C de pe host rezultatul din matricea d_C de pe device
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatrixMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  float Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  for (int e = 0; e < A.width; ++e)
     Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];


  C.elements[row * C.width + col] = Cvalue;

}
