#include "hip/hip_runtime.h"
/**
 * Tema 4 ASC - 2D Convolution
 *
 * Copyright (C) 2014, Flavius Tirnacop 331CA <flavius.tirnacop@cti.pub.ro>
 *
 * Unauthorized copying of this file, via any medium is strictly prohibited
 * Proprietary and confidential
 *
 */

/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P);

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{

    //TODO: calculul rezultatului convoluției
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum=0;
    int m,n;

    if ((row >= N.height) || (col >= N.width) || (row < 0) || (col < 0)) return;

    for (m = 0 ; m < 5 ; m++)
        for (n=0 ; n < 5 ; n++) {
                if((row+m-2 >= 0) && (row+m-2 < N.height) && (col+n-2 >= 0) && (col+n-2 < N.height))
                    sum += M.elements[m*M.width+n] * N.elements[(row+m-2) * N.width+(col+n-2)];
        }
    P.elements[row*P.width+col] = sum;

}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{

    //TODO: calculul rezultatului convoluției

}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++){
        printf("A=%d B=%d\n", A.elements[i] , B.elements[i]);
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    }
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    int width = 0, height = 0;
    FILE *f, *out;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);
    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    fscanf(f, "%d%d", &width, &height);
    Matrix M;//kernel de pe host
    Matrix N;//matrice inițială de pe host
    Matrix P;//rezultat fără memorie partajată calculat pe GPU
    Matrix PS;//rezultatul cu memorie partajată calculat pe GPU

    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);

    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);


    // M * N pe device
    ConvolutionOnDevice(M, N, P);

    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS);

    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.width, P.height);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);

    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(out, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
  //  int ress = CompareMatrices(reference, PS);
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(out, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");

    // Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    size_t size;
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //TODO: alocați matricele de pe device
    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    //TODO: copiați datele de pe host (M, N) pe device (MD, Nd)
    size = M.width * M.height * sizeof(float);
    hipMemcpy(Md.elements, M.elements, size, hipMemcpyHostToDevice);
    size = N.width * N.height * sizeof(float);
    hipMemcpy(Nd.elements, N.elements, size, hipMemcpyHostToDevice);

    //TODO: setați configurația de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N.width / dimBlock.x, N.height / dimBlock.y);

    sdkStartTimer(&kernelTime);
    //TODO: lansați în execuție kernelul
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel: %f ms\n", sdkGetTimerValue(&kernelTime));
    //TODO: copiaţi rezultatul pe host
    size = P.width * P.height * sizeof(float);
    hipMemcpy(P.elements, Pd.elements, size, hipMemcpyDeviceToHost);

    //TODO: eliberați memoria matricelor de pe device\
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P)
{
    size_t size;
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //TODO: alocați matricele de pe device
    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    //TODO: copiați datele de pe host (M, N) pe device (MD, Nd)
    size = M.width * M.height * sizeof(float);
    hipMemcpy(Md.elements, M.elements, size, hipMemcpyHostToDevice);
    size = N.width * N.height * sizeof(float);
    hipMemcpy(Nd.elements, N.elements, size, hipMemcpyHostToDevice);

    //TODO: setați configurația de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N.width / dimBlock.x, N.height / dimBlock.y);

    sdkStartTimer(&kernelTime);
    //TODO: lansați în execuție kernelul
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel cu memorie partajată: %f ms\n", sdkGetTimerValue(&kernelTime));
    //TODO: copiaţi rezultatul pe host
    size = P.width * P.height * sizeof(float);
    hipMemcpy(P.elements, Pd.elements, size, hipMemcpyDeviceToHost);

    //TODO: eliberați memoria matricelor de pe device
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


// Alocă o matrice de dimensiune height*width pe device
Matrix AllocateDeviceMatrix(int width, int height)
{
    Matrix m;
    //TODO: alocați matricea și setați width, pitch și height
    m.width = m.pitch = width;
    m.height = height;
    int size = m.width * m.height * sizeof(float);
    hipMalloc((void **) &m.elements, size);
    return m;
}

// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = (float*) malloc(size*sizeof(float));
    return M;
}

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}
